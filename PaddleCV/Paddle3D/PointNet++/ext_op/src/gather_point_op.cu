#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/platform/cuda_primitives.h"

#include "util.cu.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void GatherPointKernel(int b,
                                  int n,
                                  int m,
                                  const T *__restrict__ inp,
                                  const int *__restrict__ idx,
                                  T *__restrict__ out) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m;
         j += blockDim.x * gridDim.y) {
      int a = idx[i * m + j];
      for (int k = 0; k < 3; k++) {
        out[(i * m + j) * 3 + k] = inp[(i * n + a) * 3 + k];
      }
    }
  }
}

template <typename T>
__global__ void GatherPointGradKernel(int b,
                                      int n,
                                      int m,
                                      const T *__restrict__ out_grad,
                                      const int *__restrict__ idx,
                                      T *__restrict__ in_grad) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m;
         j += blockDim.x * gridDim.y) {
      int a = idx[i * m + j];
      const T *out_grad_pos = &out_grad[(i * m + j) * 3];
      T *in_grad_pos = &in_grad[(i * n + a) * 3];
      for (int k = 0; k < 3; k++) {
        platform::CudaAtomicAdd(&in_grad_pos[k], out_grad_pos[k]);
      }
    }
  }
}

template <typename T>
class GatherPointOpCUDAKernel : public framework::OpKernel<T> {
public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *points = ctx.Input<Tensor>("X");
    auto *index = ctx.Input<Tensor>("Index");
    auto *output = ctx.Output<Tensor>("Output");

    if (points->numel() == 0) return;

    const T *p_points = points->data<T>();
    const int *p_index = index->data<int>();
    T *p_out_points = output->mutable_data<T>(ctx.GetPlace());

    int batch_size = points->dims()[0];
    int n_points = points->dims()[1];
    int m_points = index->dims()[1];

    GatherPointKernel<<<dim3(2, 8, 1), 512>>>(
        batch_size, n_points, m_points, p_points, p_index, p_out_points);
  }
};

template <typename T>
class GatherPointGradOpCUDAKernel : public framework::OpKernel<T> {
public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    auto *points = ctx.Input<Tensor>("X");
    auto *index = ctx.Input<Tensor>("Index");
    auto *output_grad = ctx.Input<Tensor>(framework::GradVarName("Output"));
    auto *points_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    if (points->numel() == 0) return;

    const T *p_output_grad = output_grad->data<T>();
    const int *p_index = index->data<int>();
    T *p_points_grad = points_grad->mutable_data<T>(ctx.GetPlace());
    int pnum = points_grad->numel();

    auto &dev_ctx = ctx.template device_context<platform::CUDADeviceContext>();
    Zero<<<(pnum + 512 - 1) / 512, 512, 0, dev_ctx.stream()>>>(p_points_grad,
                                                               pnum);

    int batch_size = points->dims()[0];
    int n_points = points->dims()[1];
    int m_points = index->dims()[1];

    GatherPointGradKernel<<<dim3(2, 8, 1), 512, 0, dev_ctx.stream()>>>(
        batch_size, n_points, m_points, p_output_grad, p_index, p_points_grad);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(gather_point,
                        ops::GatherPointOpCUDAKernel<float>,
                        ops::GatherPointOpCUDAKernel<double>,
                        ops::GatherPointOpCUDAKernel<int>);
REGISTER_OP_CUDA_KERNEL(gather_point_grad,
                        ops::GatherPointGradOpCUDAKernel<float>,
                        ops::GatherPointGradOpCUDAKernel<double>,
                        ops::GatherPointGradOpCUDAKernel<int>);
